#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cmath>

#include <iostream>
#include <string>

#include <vector>


//const long MAXDIM = 10;
//const double RMIN = 2.0;
//const double RMAX = 8.0;

long powlong(long n, long k)
/* Evaluate n**k where both are long integers */
{
    long p = 1;
    for (long i = 0; i < k; ++i) p *= n;
    return p;
}


__global__ void cudaShoot(long* dev_array, long* index , long ndim,  long halfb,  long ntotal)
{

    const long base = 2 * halfb + 1;
    const double rsquare = radius * radius;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //long* index = new long[ndim];
    //std::cout<< "--------------------------------------------index.size==="<<ndim<<std::endl;
    //for (long i = 0; i < ndim; ++i) index[i] = 0;

    long idx = 0;
    int num = tid;
    while (num != 0) {
        long rem = num % base;
        num = num / base;
        index[idx + tid * ndim] = rem;
        ++idx;
    }

    double rtestsq = 0;
    for (long k = 0; k < ndim; ++k) {
        double xk = index[k] - halfb;
        rtestsq += xk * xk;
    }

    if (rtestsq < rsquare)
    {

        dev_array[tid] = 1;
    }

    else
    {
        dev_array[tid] = 0;
    }

}



int main(void)
{

    const long ntrials = 1;

    for (long n = 0; n < ntrials; ++n)
    {
        const double radius = 2.05;//drand48() * (RMAX - RMIN) + RMIN;
        const long  ndim = 2;//lrand48() % (MAXDIM - 1) + 1;
        std::cout << "### " << n << " " << radius << " " << ndim << " ... " << std::endl;

        const long halfb = static_cast<long>(floor(radius));
        const long ntotal = powlong(base, ndim);
        size_t size = ntotal * sizeof(long);
        size_t index_size = ntotal * ndim * sizeof(long);



        std::cout << "ntotal -> " << ntotal << " " << std::endl;
        // CUDA event types used for timing execution
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);




        // Allocate in HOST memory
        long* host_array = (long*)malloc(size);

        long* host_index = (long*)malloc(index_size);
        // Initialize vectors
        for (int i = 0; i < ntotal; ++i) {
            host_array[i] = 0;
        }

        for (int i = 0; i < ntotal * ndim; ++i) {
            host_index[i] = 0;
        }




        // Allocate in DEVICE memory
        long *dev_array, *dev_index;
        hipMalloc(&dev_array, size);
        hipMalloc(&dev_index, index_size);


        hipMemcpy(dev_array, host_array, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_index, host_index, index_size, hipMemcpyHostToDevice);


        // Set up layout of kernel grid
        int threadsPerBlock = 1024;
        int blocksPerGrid = (ntotal + threadsPerBlock - 1) / threadsPerBlock;

        std::cout << "###  blocksPerGrid ########"  << blocksPerGrid<<std::endl;

        hipEventRecord(start, 0);

        cudaShoot<<<blocksPerGrid, threadsPerBlock>>>(dev_array, dev_index, ndim, halfb, ntotal);


        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float time;  // Must be a float
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        std::cout << "Kernel took: " << time << " ms" << std::endl;


        hipMemcpy(host_array, dev_array, size, hipMemcpyDeviceToHost);


        long counter = 0;

        std::cout << "ntotal -> " << ntotal << " " << std::endl;
        for (long i=0; i< ntotal; i++)
        {
            counter += host_array[i];
        }


        std::cout << " -> " << counter << " " << std::endl;
        hipFree(dev_array);
        hipFree(dev_index);

        free(host_array);





    }
}
